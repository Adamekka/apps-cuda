#include "hip/hip_runtime.h"
#include "consts.hpp"
#include <iostream>

__global__ auto kernel_multiply(float* arr, float multiplier) -> void {
    size_t idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (idx < ARRAY_LEN)
        arr[idx] *= multiplier;
}

auto cuda_print_err(const hipError_t err) -> void {
    if (err != hipSuccess)
        std::cout << "CUDA Error [" << __LINE__ << "] - '"
                  << hipGetErrorString(err) << "'\n";
}

auto cuda_run_multiply(float* arr, const float multiplier) -> void {
    constexpr size_t THREADS = 128;
    constexpr size_t BLOCKS = (ARRAY_LEN + THREADS - 1) / THREADS;

    float* cuda_arr = nullptr;
    hipError_t err = hipMalloc(&cuda_arr, ARRAY_SIZE);
    cuda_print_err(err);

    err = hipMemcpy(cuda_arr, arr, ARRAY_SIZE, hipMemcpyHostToDevice);
    cuda_print_err(err);

    kernel_multiply<<<BLOCKS, THREADS>>>(cuda_arr, multiplier);
    cuda_print_err(hipGetLastError());

    err = hipMemcpy(arr, cuda_arr, ARRAY_SIZE, hipMemcpyDeviceToHost);
    cuda_print_err(err);

    hipFree(cuda_arr);
}
